#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define GROUP_SIZE 4

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_img, size_t pitch, int resX, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    
    
    float c_im = lowerY + thisY * stepY;
    for (int g = 0; g < GROUP_SIZE; g++) {
        float c_re = lowerX + thisX * stepX;
        float z_re = c_re, z_im = c_im;

        int i;
        for (i = 0; i < maxIterations; ++i) {

            if (z_re * z_re + z_im * z_im > 4.f)
                break;

            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = c_re + new_re;
            z_im = c_im + new_im;
        }

        *((int*)((char*)d_img + thisY * pitch) + thisX) = i;
        thisX++;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int size = resX * resY * sizeof(int);
    int *h_img, *d_img;
    
    size_t pitch;
    hipHostAlloc((void **)&h_img, size, hipHostMallocDefault);
    hipMallocPitch((void **)&d_img, &pitch, resX * sizeof(int), resY);
    
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks(resX / (BLOCK_SIZE * GROUP_SIZE), resY / BLOCK_SIZE);

    mandelKernel<<<numBlocks, threadsPerBlock>>>(lowerX, lowerY, stepX, stepY, d_img, pitch, resX, maxIterations);
    
    hipMemcpy2D(h_img, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, h_img, size);

    hipHostFree(h_img);
    hipFree(d_img);
}
